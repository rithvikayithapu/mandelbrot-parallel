#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Image dimensions
#define WIDTH 2560
#define HEIGHT 1440

// Complex plane boundaries
#define RE_MIN -2.5
#define RE_MAX 1.0
#define IM_MIN -1.2
#define IM_MAX 1.2

// Maximum iterations and escape radius
#define MAX_ITER 1000
#define ESCAPE_RADIUS 2.0
#define ESCAPE_RADIUS_SQ (ESCAPE_RADIUS * ESCAPE_RADIUS)

// Color lookup table size
#define COLOR_TABLE_SIZE 4096

// CUDA block size
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// Color structure - byte-aligned for better memory access
typedef struct {
    unsigned char r, g, b;
} __attribute__((packed)) Color;

// Get time in milliseconds
double get_time_ms() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000.0) + (tv.tv_usec / 1000.0);
}

// Check CUDA errors
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Host-side color table
Color h_color_table[COLOR_TABLE_SIZE];

// Device-side color table
__constant__ Color d_color_table[COLOR_TABLE_SIZE];

// Initialize color table (host side)
void init_color_table() {
    for (int i = 0; i < COLOR_TABLE_SIZE; i++) {
        double normalized_i = (double)i / COLOR_TABLE_SIZE;
        double hue = fmod(normalized_i * MAX_ITER * 15, 360.0);
        double saturation = 0.8;
        double value = 1.0 - 0.6 * normalized_i;
        
        // HSV to RGB conversion
        double c = value * saturation;
        double x = c * (1 - fabs(fmod(hue / 60.0, 2) - 1));
        double m = value - c;
        
        double r = 0, g = 0, b = 0;
        
        if (hue < 60) {
            r = c; g = x; b = 0;
        } else if (hue < 120) {
            r = x; g = c; b = 0;
        } else if (hue < 180) {
            r = 0; g = c; b = x;
        } else if (hue < 240) {
            r = 0; g = x; b = c;
        } else if (hue < 300) {
            r = x; g = 0; b = c;
        } else {
            r = c; g = 0; b = x;
        }
        
        h_color_table[i].r = (unsigned char)((r + m) * 255);
        h_color_table[i].g = (unsigned char)((g + m) * 255);
        h_color_table[i].b = (unsigned char)((b + m) * 255);
    }
    
    // Copy color table to device constant memory
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_color_table), h_color_table, 
                                         sizeof(Color) * COLOR_TABLE_SIZE));
}

// CUDA kernel for Mandelbrot set computation
__global__ void mandelbrot_kernel(Color* output, int width, int height,
                                  double re_min, double re_max, 
                                  double im_min, double im_max) {
    // Calculate pixel coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within image bounds
    if (x >= width || y >= height) return;
    
    // Map pixel to complex plane
    double re_scale = (re_max - re_min) / (width - 1);
    double im_scale = (im_max - im_min) / (height - 1);
    
    double cr = re_min + x * re_scale;  // real component
    double ci = im_max - y * im_scale;  // imaginary component
    
    // Initialize z = 0
    double zr = 0.0;
    double zi = 0.0;
    double zr2 = 0.0;
    double zi2 = 0.0;
    
    // Iterate until escape or max iterations
    int iter;
    for (iter = 0; iter < MAX_ITER; iter++) {
        zi = 2.0 * zr * zi + ci;
        zr = zr2 - zi2 + cr;
        zr2 = zr * zr;
        zi2 = zi * zi;
        
        // Check escape condition
        if (zr2 + zi2 > ESCAPE_RADIUS_SQ)
            break;
    }
    
    // Calculate smooth color
    double smooth_iter;
    if (iter == MAX_ITER) {
        smooth_iter = (double)MAX_ITER;
    } else {
        // Smooth coloring formula
        double mag = sqrt(zr2 + zi2);
        smooth_iter = iter + 1.0 - log(log(mag)) / log(2.0);
    }
    
    // Get color from lookup table
    Color color;
    if (smooth_iter >= MAX_ITER) {
        color = {0, 0, 0};  // Black for points in the set
    } else {
        int idx = (int)(smooth_iter * COLOR_TABLE_SIZE / MAX_ITER) % COLOR_TABLE_SIZE;
        color = d_color_table[idx];
    }
    
    // Write color to output buffer
    output[y * width + x] = color;
}

// Write the image data to a PPM file
void write_ppm(const char* filename, Color* image_data, int width, int height) {
    double start_time = get_time_ms();
    
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
        return;
    }
    
    // Write PPM header
    fprintf(fp, "P6\n%d %d\n255\n", width, height);
    
    // Write image data in one large block
    fwrite(image_data, sizeof(Color), width * height, fp);
    
    fclose(fp);
    double end_time = get_time_ms();
    printf("Image written to %s (%.2f ms)\n", filename, end_time - start_time);
}

// Print device information
void print_device_info() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    printf("Found %d CUDA device(s)\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        printf("Device %d: %s\n", i, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Global memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("  Memory clock rate: %d kHz\n", prop.memoryClockRate);
        printf("  Memory bus width: %d bits\n", prop.memoryBusWidth);
    }
    printf("\n");
}

int main(int argc, char *argv[]) {
    // Parse command line arguments for dimensions and output file
    int width = (argc > 1) ? atoi(argv[1]) : WIDTH;
    int height = (argc > 2) ? atoi(argv[2]) : HEIGHT;
    const char *outfile = (argc > 3) ? argv[3] : "mandelbrot_cuda.ppm";
    
    // Start timing
    double total_start_time = get_time_ms();
    
    // Print device information
    print_device_info();
    
    // Initialize color table
    double init_start_time = get_time_ms();
    init_color_table();
    double init_end_time = get_time_ms();
    printf("Color table initialization: %.2f ms\n", init_end_time - init_start_time);
    
    // Allocate host memory for image
    Color* h_image_data = (Color*)malloc(width * height * sizeof(Color));
    if (h_image_data == NULL) {
        fprintf(stderr, "Host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    
    // Allocate device memory for image
    Color* d_image_data;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_image_data, width * height * sizeof(Color)));
    
    // Calculate grid and block dimensions
    dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, 
                 (height + blockDim.y - 1) / blockDim.y);
    
    printf("Grid dimensions: %d x %d blocks\n", gridDim.x, gridDim.y);
    printf("Block dimensions: %d x %d threads\n", blockDim.x, blockDim.y);
    printf("Total threads: %d\n\n", gridDim.x * gridDim.y * blockDim.x * blockDim.y);
    
    // Start kernel timing
    double kernel_start_time = get_time_ms();
    
    // Launch CUDA kernel
    mandelbrot_kernel<<<gridDim, blockDim>>>(d_image_data, width, height, 
                                            RE_MIN, RE_MAX, IM_MIN, IM_MAX);
    
    // Synchronize and check for errors
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipGetLastError());
    
    double kernel_end_time = get_time_ms();
    double kernel_time = kernel_end_time - kernel_start_time;
    printf("CUDA kernel execution: %.2f ms\n", kernel_time);
    
    // Copy result back to host
    double copy_start_time = get_time_ms();
    CHECK_CUDA_ERROR(hipMemcpy(h_image_data, d_image_data, 
                              width * height * sizeof(Color), 
                              hipMemcpyDeviceToHost));
    double copy_end_time = get_time_ms();
    printf("Device to host transfer: %.2f ms\n", copy_end_time - copy_start_time);
    
    // Write the image to file
    write_ppm(outfile, h_image_data, width, height);
    
    // Clean up
    free(h_image_data);
    hipFree(d_image_data);
    
    // End timing
    double total_end_time = get_time_ms();
    double total_time = total_end_time - total_start_time;
    
    // Print performance summary
    printf("\n--- Performance Summary ---\n");
    printf("Image resolution: %d x %d (%d pixels)\n", width, height, width * height);
    printf("Maximum iterations: %d\n", MAX_ITER);
    printf("Kernel execution: %.2f ms (%.2f MPixels/sec)\n", 
           kernel_time, (width * height) / (kernel_time * 1000));
    printf("Total runtime: %.2f ms (%.2f MPixels/sec)\n", 
           total_time, (width * height) / (total_time * 1000));
    
    return 0;
}
